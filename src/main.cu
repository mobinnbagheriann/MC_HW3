#include "hip/hip_runtime.h"
#ifndef _GNU_SOURCE
#define _GNU_SOURCE  // For getline and strdup
#endif
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <pthread.h>
#include <hs/hs.h>
#include <sys/stat.h>
#include <unistd.h>

// CUDA includes for GPU mode
#include <hip/hip_runtime.h>
#include <>

// cuDF RAPIDS includes for GPU regex
#include <rmm/device_uvector.hpp>
#include <rmm/cuda_stream_view.hpp>
#include <rmm/mr/device/per_device_resource.hpp>
#include <rmm/mr/device/pool_memory_resource.hpp>
#include <cudf/column/column.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/strings/contains.hpp>
#include <cudf/strings/regex/regex_program.hpp>
#include <cudf/types.hpp>

// CUDA error checking macro with detailed error reporting
#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA Error detected. %s %s\n", hipGetErrorName(err), hipGetErrorString(err)); \
            fprintf(stderr, "CUDA error at %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

// Additional macro for checking CUDA errors after kernel launches
#define CUDA_CHECK_KERNEL() \
    do { \
        hipError_t err = hipGetLastError(); \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA Kernel Error detected. %s %s\n", hipGetErrorName(err), hipGetErrorString(err)); \
            fprintf(stderr, "CUDA kernel error at %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)


// --- Data Structures ---

// Execution Mode
typedef enum {
    MODE_CPU,
    MODE_GPU
} execution_mode_t;

// Configuration Structure
typedef struct {
    execution_mode_t mode;
    char* rules_file;
    char* input_file;
    int num_threads;  // Only used for CPU mode
} config_t;

/**
 * @struct MatchContext
 * @brief Context structure passed to the Hyperscan match event handler.
 */
typedef struct {
    int* matches;           // Array to store IDs of matched rules.
    int match_count;        // Number of matches found for the current line.
    int match_capacity;     // Allocated capacity of the matches array.
} MatchContext;

/**
 * @struct ThreadData
 * @brief Data structure to pass information to each worker thread.
 */
typedef struct {
    int thread_id;                 // Unique identifier for the thread.
    char** lines;                  // Pointer to the array of all input lines.
    unsigned int* line_lengths;    // Pointer to the array of all line lengths.
    long start_line;               // Starting line index for this thread.
    long end_line;                 // Ending line index for this thread.
    hs_database_t* database;       // Pointer to the compiled Hyperscan database.
    hs_scratch_t* scratch;         // Per-thread scratch space for Hyperscan.
    char*** thread_results;        // 2D array: [line_index][match_list] for this thread's lines
    long total_matches;            // Total number of matches found by this thread.
} ThreadData;


// --- Forward Declarations ---
int run_cpu_mode(const config_t* config);
int run_gpu_mode(const config_t* config);

// cuDF helper functions for GPU mode
#ifdef __cplusplus
extern "C" {
#endif

// Build device strings column from host vector<string>
static std::unique_ptr<cudf::column>
make_device_strings(const std::vector<std::string>& h, rmm::cuda_stream_view stream) {
    using size_type = cudf::size_type;
    const size_type n = static_cast<size_type>(h.size());

    // Handle edge case of empty input
    if (n == 0) {
        return cudf::make_empty_column(cudf::data_type{cudf::type_id::STRING});
    }

    std::vector<int32_t> h_offsets(n + 1, 0);
    size_t total_chars = 0;
    for (size_t i = 0; i < h.size(); ++i) {
        total_chars += h[i].size();
        h_offsets[i + 1] = static_cast<int32_t>(total_chars);
    }
    
    std::vector<char> h_chars;
    h_chars.reserve(total_chars);
    for (auto& s : h) h_chars.insert(h_chars.end(), s.begin(), s.end());

    // Allocate device memory with explicit error checking
    rmm::device_uvector<int32_t> d_offsets(n + 1, stream);
    rmm::device_uvector<char> d_chars(total_chars, stream);

    CUDA_CHECK(hipMemcpyAsync(d_offsets.data(), h_offsets.data(),
                               (n + 1) * sizeof(int32_t),
                               hipMemcpyHostToDevice, stream.value()));
    if (total_chars > 0) {
        CUDA_CHECK(hipMemcpyAsync(d_chars.data(), h_chars.data(), total_chars,
                                   hipMemcpyHostToDevice, stream.value()));
    }
    
    // Synchronize to ensure data transfer is complete
    CUDA_CHECK(hipStreamSynchronize(stream.value()));

    auto null_mask = rmm::device_buffer{0, stream};
    cudf::size_type null_count = 0;

    auto offsets_buf = d_offsets.release();
    auto offsets_col = std::make_unique<cudf::column>(
        cudf::data_type{cudf::type_id::INT32},
        n + 1,
        std::move(offsets_buf),
        rmm::device_buffer{0, stream},
        0);
    auto chars_buf = d_chars.release();
    return cudf::make_strings_column(
        n,
        std::move(offsets_col),
        std::move(chars_buf),
        null_count,
        std::move(null_mask));
}

__global__ void add_true_to_counts(const uint8_t* __restrict__ vals,
                                   int n,
                                   int* __restrict__ counts) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) counts[i] += (vals[i] != 0);
}

#ifdef __cplusplus
}
#endif

// --- Utility Functions ---

/**
 * @brief Prints an error message and exits the program.
 */
void fail(const char* msg) {
    fprintf(stderr, "ERROR: %s\n", msg);
    exit(EXIT_FAILURE);
}

/**
 * @brief Print usage information.
 */
void print_usage(const char* program_name) {
    printf("Usage: %s --mode <cpu|gpu> --rules <rules_file> --input <input_file> [--threads <num_threads>]\n", program_name);
    printf("\nRequired arguments:\n");
    printf("  --mode      <cpu|gpu>       Processing mode (CPU or GPU)\n");
    printf("  --rules     <rules_file>    Path to the rules file\n");
    printf("  --input     <input_file>    Path to the input file\n");
    printf("\nOptional arguments:\n");
    printf("  --threads   <num_threads>   Number of threads (required for CPU mode)\n");
    printf("\nOutput files are automatically generated in the results/ directory:\n");
    printf("  Results_HW3_MCC_030402_401110686_{CPU/GPU}_{DataSet}_{NumThreads/Library}.txt\n");
    printf("  Results_HW3_MCC_030402_401110686_{CPU/GPU}_{DataSet}_{Hyperscan/GPULibrary}.csv\n");
    printf("\nExample:\n");
    printf("  %s --mode cpu --rules rules.txt --input set1.txt --threads 4\n", program_name);
    printf("  %s --mode gpu --rules rules.txt --input set1.txt\n", program_name);
    exit(EXIT_FAILURE);
}

/**
 * @brief Generate automatic output filename based on configuration.
 */
char* generate_output_filename(const config_t* config) {
    // Extract dataset name from input file (e.g., "set1.txt" -> "set1")
    const char* input_basename = strrchr(config->input_file, '/');
    if (input_basename) {
        input_basename++; // Skip the '/'
    } else {
        input_basename = config->input_file;
    }
    
    // Remove file extension
    char dataset[256];
    strncpy(dataset, input_basename, sizeof(dataset) - 1);
    dataset[sizeof(dataset) - 1] = '\0';
    char* dot = strrchr(dataset, '.');
    if (dot) {
        *dot = '\0';
    }
    
    // Allocate memory for the filename
    char* filename = (char*)malloc(512);
    if (!filename) {
        fprintf(stderr, "Error: Memory allocation failed for output filename\n");
        exit(EXIT_FAILURE);
    }
    
    if (config->mode == MODE_CPU) {
        snprintf(filename, 512, "results/Results_HW3_MCC_030402_401110686_CPU_%s_%d.txt", 
                 dataset, config->num_threads);
    } else {
        snprintf(filename, 512, "results/Results_HW3_MCC_030402_401110686_GPU_%s_CUDA.txt", 
                 dataset);
    }
    
    return filename;
}

/**
 * @brief Generate performance CSV filename based on configuration.
 */
char* generate_performance_filename(const config_t* config, const char* input_filename) {
    // Extract dataset name from input filename
    const char* dataset_name = strrchr(input_filename, '/');
    if (dataset_name) {
        dataset_name++; // Skip the '/'
    } else {
        dataset_name = input_filename;
    }
    
    // Remove extension from dataset name
    char* dataset_clean = strdup(dataset_name);
    char* dot = strrchr(dataset_clean, '.');
    if (dot) *dot = '\0';
    
    char* filename = (char*)malloc(512);
    if (config->mode == MODE_CPU) {
        snprintf(filename, 512, "results/Results_HW3_MCC_030402_401110686_CPU_%s_Hyperscan.csv", 
                 dataset_clean);
    } else {
        snprintf(filename, 512, "results/Results_HW3_MCC_030402_401110686_GPU_%s_CUDA.csv", 
                 dataset_clean);
    }
    
    free(dataset_clean);
    return filename;
}

/**
 * @brief Parse command line arguments.
 */
config_t parse_arguments(int argc, char* argv[]) {
    config_t config{};
    
    if (argc < 5) {  // Minimum required arguments for GPU mode
        print_usage(argv[0]);
    }
    
    for (int i = 1; i < argc; i++) {
        if (strcmp(argv[i], "--mode") == 0 && i + 1 < argc) {
            if (strcmp(argv[i + 1], "cpu") == 0) {
                config.mode = MODE_CPU;
            } else if (strcmp(argv[i + 1], "gpu") == 0) {
                config.mode = MODE_GPU;
            } else {
                fprintf(stderr, "ERROR: Invalid mode '%s'. Use 'cpu' or 'gpu'.\n", argv[i + 1]);
                print_usage(argv[0]);
            }
            i++; // Skip next argument
        } else if (strcmp(argv[i], "--rules") == 0 && i + 1 < argc) {
            config.rules_file = argv[i + 1];
            i++;
        } else if (strcmp(argv[i], "--input") == 0 && i + 1 < argc) {
            config.input_file = argv[i + 1];
            i++;
        } else if (strcmp(argv[i], "--threads") == 0 && i + 1 < argc) {
            config.num_threads = atoi(argv[i + 1]);
            if (config.num_threads <= 0) {
                fprintf(stderr, "ERROR: Number of threads must be a positive integer.\n");
                print_usage(argv[0]);
            }
            i++;
        }
    }
    
    // Validate required arguments
    if (!config.rules_file || !config.input_file) {
        fprintf(stderr, "ERROR: Missing required arguments.\n");
        print_usage(argv[0]);
    }
    
    if (config.mode == MODE_CPU && config.num_threads == 0) {
        fprintf(stderr, "ERROR: --threads argument is required for CPU mode.\n");
        print_usage(argv[0]);
    }
    
    return config;
}

/**
 * @brief Reads all lines from a file into a dynamically allocated array.
 */
char** read_lines_from_file(const char* filename, long* line_count, unsigned int** line_lengths, long* total_bytes) {
    FILE* file = fopen(filename, "r");
    if (!file) {
        perror("fopen failed");
        fail("Could not open file.");
    }

    // Get file size for total_bytes metric
    struct stat st;
    if (stat(filename, &st) == 0) {
        *total_bytes = st.st_size;
    } else {
        *total_bytes = 0; // Fallback
    }

    long capacity = 1024;
    char** lines = (char**)malloc(capacity * sizeof(char*));
    if (!lines) fail("Failed to allocate memory for lines.");

    *line_count = 0;
    char* line_buffer = NULL;
    size_t buffer_size = 0;

    while (getline(&line_buffer, &buffer_size, file) != -1) {
        if (*line_count >= capacity) {
            capacity *= 2;
            lines = (char**)realloc(lines, capacity * sizeof(char*));
            if (!lines) fail("Failed to reallocate memory for lines.");
        }
        // Strip newline characters
        line_buffer[strcspn(line_buffer, "\r\n")] = 0;
        lines[*line_count] = strdup(line_buffer);
        if (!lines[*line_count]) fail("Failed to duplicate line.");
        (*line_count)++;
    }

    free(line_buffer);
    fclose(file);

    // Create the line lengths array
    *line_lengths = (unsigned int*)malloc(*line_count * sizeof(unsigned int));
    if (!*line_lengths) fail("Failed to allocate memory for line lengths.");
    for (long i = 0; i < *line_count; i++) {
        (*line_lengths)[i] = strlen(lines[i]);
    }

    return lines;
}

// --- Hyperscan Match Callback ---

/**
 * @brief Hyperscan match event handler.
 */
static int onMatch(unsigned int id, unsigned long long from, unsigned long long to,
                   unsigned int flags, void* ctx) {
    (void)from;   // Suppress unused parameter warning
    (void)to;     // Suppress unused parameter warning
    (void)flags;  // Suppress unused parameter warning
    
    MatchContext* context = (MatchContext*)ctx;

    // Resize matches array if needed
    if (context->match_count >= context->match_capacity) {
        context->match_capacity *= 2;
        context->matches = (int*)realloc(context->matches, context->match_capacity * sizeof(int));
        if (!context->matches) {
            fail("Failed to reallocate memory for matches in callback.");
        }
    }

    context->matches[context->match_count++] = id;
    return 0; // Continue scanning
}


// --- Worker Thread ---

/**
 * @brief The main function for each worker thread.
 */
void* worker_thread(void* arg) {
    ThreadData* data = (ThreadData*)arg;
    data->total_matches = 0;

    // Allocate scratch space for this thread
    hs_error_t scratch_err = hs_alloc_scratch(data->database, &data->scratch);
    if (scratch_err != HS_SUCCESS) {
        fprintf(stderr, "Thread %d: Failed to allocate scratch space. Error: %d\n", data->thread_id, scratch_err);
        return NULL;
    }

    // Allocate 2D result array for this thread's lines
    long thread_line_count = data->end_line - data->start_line;
    data->thread_results = (char***)malloc(thread_line_count * sizeof(char**));
    if (!data->thread_results) {
        fprintf(stderr, "Thread %d: Failed to allocate thread results array.\n", data->thread_id);
        return NULL;
    }

    for (long i = data->start_line; i < data->end_line; i++) {
        long local_index = i - data->start_line; // Local index within this thread's range
        
        // Initialize context for this line's scan
        MatchContext context;
        context.match_capacity = 16; // Initial capacity
        context.matches = (int*)malloc(context.match_capacity * sizeof(int));
        if (!context.matches) {
             data->thread_results[local_index] = (char**)malloc(sizeof(char*));
             data->thread_results[local_index][0] = strdup(""); // Store empty result on failure
             continue;
        }
        context.match_count = 0;

        // Perform the scan
        hs_error_t err = hs_scan(data->database, data->lines[i], data->line_lengths[i], 0,
                                 data->scratch, onMatch, &context);

        if (err != HS_SUCCESS) {
            free(context.matches);
            data->thread_results[local_index] = (char**)malloc(sizeof(char*));
            data->thread_results[local_index][0] = strdup(""); // Store empty result on error
            continue;
        }

        data->total_matches += context.match_count;

        // Format the result string with ZERO-INDEXED pattern numbers (e.g., "0,3,9")
        if (context.match_count > 0) {
            // A rough estimation for buffer size: 10 chars per match ID + commas
            size_t buffer_size = context.match_count * 10;
            char* result_buffer = (char*)malloc(buffer_size);
            if (!result_buffer) {
                data->thread_results[local_index] = (char**)malloc(sizeof(char*));
                data->thread_results[local_index][0] = strdup("");
            } else {
                int offset = 0;
                for (int j = 0; j < context.match_count; j++) {
                    // Use ZERO-INDEXED pattern numbers (Hyperscan IDs start from 0)
                    offset += snprintf(result_buffer + offset, buffer_size - offset,
                                       "%d%s", context.matches[j], (j == context.match_count - 1) ? "" : ",");
                }
                data->thread_results[local_index] = (char**)malloc(sizeof(char*));
                data->thread_results[local_index][0] = result_buffer;
            }
        } else {
            // If no matches, store an empty string
            data->thread_results[local_index] = (char**)malloc(sizeof(char*));
            data->thread_results[local_index][0] = strdup("");
        }

        free(context.matches);
    }

    // Free scratch space allocated by this thread
    if (data->scratch) {
        hs_free_scratch(data->scratch);
    }

    return NULL;
}


// --- CPU Mode Implementation ---

int run_cpu_mode(const config_t* config) {
    // --- 1. Read and Compile Rules ---
    printf("Reading and compiling regex rules from '%s'...\n", config->rules_file);
    long pattern_count = 0;
    long ignored_total_bytes;
    unsigned int* ignored_lengths;
    char** patterns = read_lines_from_file(config->rules_file, &pattern_count, &ignored_lengths, &ignored_total_bytes);
    free(ignored_lengths);

    unsigned int* ids = (unsigned int*)malloc(pattern_count * sizeof(unsigned int));
    unsigned int* flags = (unsigned int*)malloc(pattern_count * sizeof(unsigned int));
    if (!ids || !flags) fail("Failed to allocate memory for rule IDs/flags.");

    for (long i = 0; i < pattern_count; i++) {
        ids[i] = i; // Hyperscan uses 0-indexed IDs
        flags[i] = 0; // No flags
    }

    hs_database_t* database;
    hs_compile_error_t* compile_err;
    hs_platform_info_t platform;
    
    // Populate platform information for optimal compilation
    hs_error_t platform_err = hs_populate_platform(&platform);
    if (platform_err != HS_SUCCESS) {
        printf("Warning: Could not populate platform info, using default settings.\n");
    }
    
    hs_error_t err = hs_compile_multi((const char* const*)patterns, flags, ids, pattern_count,
                                      HS_MODE_BLOCK, (platform_err == HS_SUCCESS) ? &platform : NULL, 
                                      &database, &compile_err);

    if (err != HS_SUCCESS) {
        fprintf(stderr, "ERROR: Unable to compile pattern: %s\n", compile_err->message);
        hs_free_compile_error(compile_err);
        fail("Hyperscan compilation failed.");
    }
    
    if (!database) {
        fail("Database compilation succeeded but database is NULL.");
    }
    
    printf("Compilation successful. %ld rules loaded.\n", pattern_count);

    // --- 2. Read Input Data ---
    printf("Reading input data from '%s'...\n", config->input_file);
    long line_count = 0;
    long total_bytes = 0;
    unsigned int* line_lengths;
    char** lines = read_lines_from_file(config->input_file, &line_count, &line_lengths, &total_bytes);
    printf("Read %ld lines, total size: %.2f MB.\n", line_count, (double)total_bytes / (1024 * 1024));

    // --- 3. Setup and Run Threads ---
    printf("Processing with %d worker thread(s)...\n", config->num_threads);
    pthread_t* threads = (pthread_t*)malloc(config->num_threads * sizeof(pthread_t));
    ThreadData* thread_data = (ThreadData*)malloc(config->num_threads * sizeof(ThreadData));
    if (!threads || !thread_data) {
        fail("Failed to allocate memory for thread management.");
    }

    struct timespec start_time, end_time;
    clock_gettime(CLOCK_MONOTONIC, &start_time);

    long lines_per_thread = line_count / config->num_threads;
    long remaining_lines = line_count % config->num_threads;
    long current_line = 0;

    for (int i = 0; i < config->num_threads; i++) {
        thread_data[i].thread_id = i;
        thread_data[i].lines = lines;
        thread_data[i].line_lengths = line_lengths;
        thread_data[i].database = database;
        thread_data[i].thread_results = NULL; // Will be allocated by each thread
        thread_data[i].total_matches = 0;
        thread_data[i].scratch = NULL; // Let each thread allocate its own scratch

        // Distribute lines
        thread_data[i].start_line = current_line;
        long chunk_size = lines_per_thread + (i < remaining_lines ? 1 : 0);
        thread_data[i].end_line = current_line + chunk_size;
        current_line += chunk_size;

        pthread_create(&threads[i], NULL, worker_thread, &thread_data[i]);
    }

    // --- 4. Join Threads and Collect Results ---
    long total_matches = 0;
    for (int i = 0; i < config->num_threads; i++) {
        pthread_join(threads[i], NULL);
        total_matches += thread_data[i].total_matches;
    }

    // --- 5. Merge Thread Results into Final Output Array ---
    char** all_results = (char**)malloc(line_count * sizeof(char*));
    if (!all_results) {
        fail("Failed to allocate memory for final results.");
    }

    // Copy results from each thread's 2D array to the final output array
    for (int i = 0; i < config->num_threads; i++) {
        long thread_line_count = thread_data[i].end_line - thread_data[i].start_line;
        for (long j = 0; j < thread_line_count; j++) {
            long global_index = thread_data[i].start_line + j;
            all_results[global_index] = strdup(thread_data[i].thread_results[j][0]);
            
            // Free the thread's result memory
            free(thread_data[i].thread_results[j][0]);
            free(thread_data[i].thread_results[j]);
        }
        free(thread_data[i].thread_results);
    }

    clock_gettime(CLOCK_MONOTONIC, &end_time);
    printf("Processing completed.\n");

    // --- 6. Calculate Performance Metrics ---
    double elapsed_seconds = (end_time.tv_sec - start_time.tv_sec) +
                             (end_time.tv_nsec - start_time.tv_nsec) / 1e9;

    double throughput_input_per_sec = line_count / elapsed_seconds;
    double throughput_mbytes_per_sec = (total_bytes / (1024.0 * 1024.0)) / elapsed_seconds;
    double throughput_match_per_sec = total_matches / elapsed_seconds;
    double latency_ms = (elapsed_seconds * 1000.0) / line_count;

    printf("Performance Metrics:\n");
    printf("  Total Time: %.4f seconds\n", elapsed_seconds);
    printf("  Total Matches: %ld\n", total_matches);
    printf("  Throughput (Input/sec): %.2f\n", throughput_input_per_sec);
    printf("  Throughput (MBytes/sec): %.2f\n", throughput_mbytes_per_sec);
    printf("  Throughput (Match/sec): %.2f\n", throughput_match_per_sec);
    printf("  Latency (ms/input): %.4f\n", latency_ms);

    // --- 7. Write Output Files ---
    char* output_filename = generate_output_filename(config);
    printf("Writing results to '%s'...\n", output_filename);

    // Write match results
    FILE* out_file = fopen(output_filename, "w");
    if (!out_file) fail("Could not open output file for writing.");
    for (long i = 0; i < line_count; i++) {
        fprintf(out_file, "%s\n", all_results[i]);
    }
    fclose(out_file);

    // Write performance metrics
    char* perf_filename = generate_performance_filename(config, config->input_file);
    FILE* perf_file = fopen(perf_filename, "a");
    if (!perf_file) fail("Could not open performance file for writing.");

    // Check if file is empty (new file) to write header
    fseek(perf_file, 0, SEEK_END);
    long file_size = ftell(perf_file);
    if (file_size == 0) {
        // File is empty, write header
        fprintf(perf_file, "threads,throughput_input_per_sec,throughput_mbytes_per_sec,throughput_match_per_sec,latency_ms\n");
    }
    
    fprintf(perf_file, "%d,%.2f,%.2f,%.2f,%.4f\n",
            config->num_threads,
            throughput_input_per_sec,
            throughput_mbytes_per_sec,
            throughput_match_per_sec,
            latency_ms);
    fclose(perf_file);
    
    printf("Results written to '%s' and '%s'\n\n", output_filename, perf_filename);
    free(output_filename);
    free(perf_filename);

    // --- 8. Cleanup ---
    hs_free_database(database);
    for (long i = 0; i < pattern_count; i++) free(patterns[i]);
    free(patterns);
    free(ids);
    free(flags);
    for (long i = 0; i < line_count; i++) {
        free(lines[i]);
        free(all_results[i]);
    }
    free(lines);
    free(line_lengths);
    free(all_results);
    free(threads);
    free(thread_data);

    return EXIT_SUCCESS;
}


// --- GPU Mode Implementation ---

int run_gpu_mode(const config_t* config) {
    printf("Starting GPU mode processing with cuDF/RAPIDS...\n");
    
    // Store memory resources to keep them in scope
    std::shared_ptr<rmm::mr::cuda_memory_resource> cuda_mr;
    std::shared_ptr<rmm::mr::pool_memory_resource<rmm::mr::cuda_memory_resource>> pool_mr;
    
    try {
        // --- 1. Initialize CUDA and RMM ---
        int device_count;
        CUDA_CHECK(hipGetDeviceCount(&device_count));
        if (device_count == 0) {
            fail("No CUDA-capable devices found");
        }
        
        // Set CUDA device explicitly
        CUDA_CHECK(hipSetDevice(0));
        
        // Initialize RMM with pool memory resource to reduce allocation overhead
        cuda_mr = std::make_shared<rmm::mr::cuda_memory_resource>();
        pool_mr = std::make_shared<rmm::mr::pool_memory_resource<rmm::mr::cuda_memory_resource>>(
            cuda_mr.get(), 1024 * 1024 * 512); // 512MB initial pool
        rmm::mr::set_current_device_resource(pool_mr.get());
        
        printf("Initialized RMM pool memory resource with 512MB initial pool\n");
        
        hipDeviceProp_t device_prop;
        CUDA_CHECK(hipGetDeviceProperties(&device_prop, 0));
        printf("Using GPU: %s\n", device_prop.name);
        
        // --- 2. Read and prepare patterns ---
        printf("Reading regex patterns from '%s'...\n", config->rules_file);
        long pattern_count = 0;
        long ignored_total_bytes;
        unsigned int* ignored_lengths;
        char** patterns = read_lines_from_file(config->rules_file, &pattern_count, &ignored_lengths, &ignored_total_bytes);
        free(ignored_lengths);
        printf("Loaded %ld patterns.\n", pattern_count);
        
        // Convert C-style string array to C++ vector for cuDF
        std::vector<std::string> pattern_vector;
        pattern_vector.reserve(pattern_count);
        for (long i = 0; i < pattern_count; i++) {
            pattern_vector.emplace_back(patterns[i]);
        }
        
        // --- 3. Read input data ---
        printf("Reading input data from '%s'...\n", config->input_file);
        long line_count = 0;
        long total_bytes = 0;
        unsigned int* line_lengths;
        char** lines = read_lines_from_file(config->input_file, &line_count, &line_lengths, &total_bytes);
        printf("Read %ld lines, total size: %.2f MB.\n", line_count, (double)total_bytes / (1024 * 1024));
        
        // Convert C-style string array to C++ vector for cuDF
        std::vector<std::string> sentence_vector;
        sentence_vector.reserve(line_count);
        for (long i = 0; i < line_count; i++) {
            sentence_vector.emplace_back(lines[i]);
        }
        
        // --- 4. Create CUDA events for precise timing ---
        hipEvent_t start_event, end_event;
        CUDA_CHECK(hipEventCreate(&start_event));
        CUDA_CHECK(hipEventCreate(&end_event));
        
        // Start timing (including data transfer)
        CUDA_CHECK(hipEventRecord(start_event, 0));
        
        // --- 5. Setup cuDF/RAPIDS processing with better memory management ---
        printf("Initializing cuDF processing...\n");
        
        // Create explicit CUDA stream for better control
        hipStream_t gpu_stream;
        CUDA_CHECK(hipStreamCreate(&gpu_stream));
        auto stream = rmm::cuda_stream_view{gpu_stream};
        
        // Force CUDA synchronization before creating columns
        CUDA_CHECK(hipDeviceSynchronize());
        
        auto sentences_col = make_device_strings(sentence_vector, stream);
        cudf::strings_column_view sview{sentences_col->view()};
        const int nrows = static_cast<int>(sview.size());
        
        // Synchronize after column creation
        CUDA_CHECK(hipStreamSynchronize(stream.value()));
        
        // Allocate device memory for match counts with explicit error checking
        rmm::device_uvector<int> d_counts(nrows, stream);
        CUDA_CHECK(hipMemsetAsync(d_counts.data(), 0, nrows * sizeof(int), stream.value()));
        CUDA_CHECK(hipStreamSynchronize(stream.value()));
        
        // --- 6. Process patterns in batches to avoid memory issues ---
        printf("Processing patterns with GPU regex matching...\n");
        long total_matches = 0;
        
        // Store all match results for each line
        std::vector<std::vector<int>> line_matches(line_count);
        
        // Process patterns in smaller batches to avoid memory pressure
        const long batch_size = 50; // Process 50 patterns at a time to be more conservative
        long num_batches = (pattern_count + batch_size - 1) / batch_size;
        
        printf("Processing %ld patterns in %ld batches of up to %ld patterns each...\n", 
               pattern_count, num_batches, batch_size);
        
        for (long batch = 0; batch < num_batches; batch++) {
            long start_idx = batch * batch_size;
            long end_idx = std::min(start_idx + batch_size, pattern_count);
            
            printf("Processing batch %ld/%ld (patterns %ld-%ld)...\n", 
                   batch + 1, num_batches, start_idx, end_idx - 1);
            
            for (long pat_idx = start_idx; pat_idx < end_idx; pat_idx++) {
                const auto& pattern = pattern_vector[pat_idx];
                
                // Skip empty or very long patterns that might cause issues
                if (pattern.empty() || pattern.length() > 1000) {
                    printf("Warning: Skipping pattern %ld (empty or too long: %zu chars)\n", 
                           pat_idx, pattern.length());
                    continue;
                }
                
                try {
                    // Create regex program for this pattern with error checking
                    auto prog = cudf::strings::regex_program::create(pattern);
                    auto bool_col = cudf::strings::contains_re(sview, *prog);
                    
                    // Ensure operations are completed before accessing results
                    CUDA_CHECK(hipStreamSynchronize(stream.value()));
                    
                    auto bv = bool_col->view();
                    const uint8_t* d_vals = bv.data<uint8_t>();
                    
                    // Verify pointer validity before kernel launch
                    if (d_vals == nullptr) {
                        printf("Warning: Pattern %ld resulted in null data pointer, skipping\n", pat_idx);
                        continue;
                    }
                    
                    // Copy boolean results back to host to track which lines matched
                    std::vector<uint8_t> h_matches(nrows);
                    CUDA_CHECK(hipMemcpyAsync(h_matches.data(), d_vals, nrows * sizeof(uint8_t),
                                               hipMemcpyDeviceToHost, stream.value()));
                    CUDA_CHECK(hipStreamSynchronize(stream.value()));
                    
                    // Record which lines matched this pattern
                    long pattern_matches = 0;
                    for (int i = 0; i < nrows; i++) {
                        if (h_matches[i] != 0) {
                            line_matches[i].push_back((int)pat_idx);
                            total_matches++;
                            pattern_matches++;
                        }
                    }
                    
                    // Print progress for complex patterns
                    if (pattern_matches > 10000 || pat_idx % 500 == 0) {
                        printf("Pattern %ld: %ld matches\n", pat_idx, pattern_matches);
                    }
                    
                    // Force cleanup of temporary objects before next iteration
                    prog.reset();
                    bool_col.reset();
                    
                } catch (const std::exception& e) {
                    printf("Warning: Failed to process pattern %ld: '%s' - Error: %s\n", 
                           pat_idx, pattern.c_str(), e.what());
                    // Synchronize stream after exception to clean up any partial operations
                    CUDA_CHECK(hipStreamSynchronize(stream.value()));
                    continue;
                }
            }
            
            // Force garbage collection and memory cleanup between batches
            CUDA_CHECK(hipStreamSynchronize(stream.value()));
            CUDA_CHECK(hipDeviceSynchronize());
            
            // Print progress
            if (batch % 10 == 0 || batch == num_batches - 1) {
                printf("Completed %ld/%ld batches, found %ld matches so far\n", 
                       batch + 1, num_batches, total_matches);
            }
        }
        
        // --- 7. Wait for all GPU operations to complete ---
        CUDA_CHECK(hipStreamSynchronize(stream.value()));
        
        // Stop timing
        CUDA_CHECK(hipEventRecord(end_event, 0));
        CUDA_CHECK(hipEventSynchronize(end_event));
        
        // Calculate elapsed time
        float elapsed_ms;
        CUDA_CHECK(hipEventElapsedTime(&elapsed_ms, start_event, end_event));
        double elapsed_seconds = elapsed_ms / 1000.0;
        
        printf("GPU processing completed.\n");
        
        // --- 8. Format results same as CPU mode ---
        printf("Formatting results...\n");
        char** all_results = (char**)malloc(line_count * sizeof(char*));
        if (!all_results) {
            fail("Failed to allocate memory for final results.");
        }
        
        for (long i = 0; i < line_count; i++) {
            const auto& matches = line_matches[i];
            if (!matches.empty()) {
                // Calculate buffer size needed
                size_t buffer_size = matches.size() * 10; // rough estimate
                char* result_buffer = (char*)malloc(buffer_size);
                if (!result_buffer) {
                    all_results[i] = strdup("");
                    continue;
                }
                
                // Build comma-separated list of pattern IDs (0-indexed like CPU mode)
                int offset = 0;
                for (size_t j = 0; j < matches.size(); j++) {
                    offset += snprintf(result_buffer + offset, buffer_size - offset,
                                       "%d%s", matches[j], (j == matches.size() - 1) ? "" : ",");
                }
                all_results[i] = result_buffer;
            } else {
                all_results[i] = strdup("");
            }
        }
        
        // --- 9. Calculate performance metrics ---
        double throughput_input_per_sec = line_count / elapsed_seconds;
        double throughput_mbytes_per_sec = (total_bytes / (1024.0 * 1024.0)) / elapsed_seconds;
        double throughput_match_per_sec = total_matches / elapsed_seconds;
        double latency_ms = (elapsed_seconds * 1000.0) / line_count;
        
        printf("Performance Metrics:\n");
        printf("  Total Time: %.4f seconds\n", elapsed_seconds);
        printf("  Total Matches: %ld\n", total_matches);
        printf("  Throughput (Input/sec): %.2f\n", throughput_input_per_sec);
        printf("  Throughput (MBytes/sec): %.2f\n", throughput_mbytes_per_sec);
        printf("  Throughput (Match/sec): %.2f\n", throughput_match_per_sec);
        printf("  Latency (ms/input): %.4f\n", latency_ms);
        
        // --- 10. Write output files ---
        char* output_filename = generate_output_filename(config);
        printf("Writing results to '%s'...\n", output_filename);
        
        // Write match results (same format as CPU mode)
        FILE* out_file = fopen(output_filename, "w");
        if (!out_file) fail("Could not open output file for writing.");
        for (long i = 0; i < line_count; i++) {
            fprintf(out_file, "%s\n", all_results[i]);
        }
        fclose(out_file);
        
        // Write performance metrics
        char* perf_filename = generate_performance_filename(config, config->input_file);
        FILE* perf_file = fopen(perf_filename, "a");
        if (!perf_file) fail("Could not open performance file for writing.");
        
        // Check if file is empty (new file) to write header
        fseek(perf_file, 0, SEEK_END);
        long file_size = ftell(perf_file);
        if (file_size == 0) {
            // File is empty, write header for GPU mode
            fprintf(perf_file, "matcher_name,throughput_input_per_sec,throughput_mbytes_per_sec,throughput_match_per_sec,latency_ms\n");
        }
        
        fprintf(perf_file, "cuDF-RAPIDS,%.2f,%.2f,%.2f,%.4f\n",
                throughput_input_per_sec,
                throughput_mbytes_per_sec,
                throughput_match_per_sec,
                latency_ms);
        fclose(perf_file);
        
        printf("Results written to '%s' and '%s'\n\n", output_filename, perf_filename);
        
        // --- 11. Cleanup ---
        // Cleanup CUDA stream first
        CUDA_CHECK(hipStreamDestroy(gpu_stream));
        
        // Cleanup CUDA events
        hipEventDestroy(start_event);
        hipEventDestroy(end_event);
        
        // Final device synchronization before freeing host memory
        CUDA_CHECK(hipDeviceSynchronize());
        
        // Reset RMM to default before cleanup
        rmm::mr::set_current_device_resource(cuda_mr.get());
        pool_mr.reset();
        cuda_mr.reset();
        
        for (long i = 0; i < pattern_count; i++) free(patterns[i]);
        free(patterns);
        for (long i = 0; i < line_count; i++) {
            free(lines[i]);
            free(all_results[i]);
        }
        free(lines);
        free(line_lengths);
        free(all_results);
        free(output_filename);
        free(perf_filename);
        
        return EXIT_SUCCESS;
        
    } catch (const std::exception& e) {
        fprintf(stderr, "GPU mode error: %s\n", e.what());
        // Cleanup memory resources on error path too
        if (pool_mr) {
            rmm::mr::set_current_device_resource(cuda_mr.get());
            pool_mr.reset();
        }
        if (cuda_mr) cuda_mr.reset();
        return EXIT_FAILURE;
    }
}


// --- Main Function ---

int main(int argc, char* argv[]) {
    config_t config = parse_arguments(argc, argv);
    
    printf("High-Performance Regex Matching - Mode: %s\n", 
           config.mode == MODE_CPU ? "CPU" : "GPU");
    
    if (config.mode == MODE_CPU) {
        return run_cpu_mode(&config);
    } else {
        return run_gpu_mode(&config);
    }
}